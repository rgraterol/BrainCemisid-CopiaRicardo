#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "utilCuda.h"
#include "timer.h"
#include "lock.h"
#include "../Class/interface.h"
#include "../Class/neuralNetwork.h"
#include "../Class/statistic.h"
#include "../Class/relationNetwork.h"
#include "../Class/culturalNet.h"
#include "../Class/countNetwork.h"


#define  RATIO               22
#define  MIN_RATIO           2
#define  TAMWORD             5

//cuda kernel prototypes
__global__ void  correct  ( unsigned char * d_vectorFlags , unsigned char * d_ptr, unsigned char *d_desiredOutput);

__global__ void  reset    ( unsigned char * d_vectorFlags , unsigned char *d_ptr);

__global__ void  recognize( unsigned char * d_vectorNeuron, unsigned char *d_vectorFlags,
                            unsigned char * d_pattern     , int *d_countHit, unsigned char *d_ptr,
                            unsigned char * d_arrayCategory  , unsigned char* d_idsNeuron,Lock lock);

__global__ void safeRelation(unsigned char *d_vectorFlags, int *d_countHit,

                             unsigned char *d_idsNeuron, unsigned char *d_vectorEar,
                             unsigned char *d_vectorSigth, stateNeuralNetwork *stateSense,
                             unsigned char *d_wishCategory);
__global__ void newItemCulturalNet(unsigned char * d_data, bool d_valve, bool trueKNW);

__global__ void  findOrderNeuron(unsigned char * d_orderNeuron, unsigned char * sightID, int * min_idx);


// methods prototype
template<class T>
inline bool equal(T a, T b);

template<class T>
bool compare(T array [] , int sizeArray);

void calculateStatistic(const float & currentTime, Statistic * & statistic, kernels kernel);
void debugTimer( GpuTimer timer);

//--------------------------------------Metodos Main-------------------------------
extern "C"
void boot(NeuralNetwork * & neuralSenses,const SizeNet & sizeNet, Statistic * & statistic){
    unsigned char * d_vectorZero;
    GpuTimer timer;

    // It allocates memory on the device
    checkCudaErrors(hipMalloc(&d_vectorZero,sizeNet.sizeVectorNeuron));

    // initialize the memory block to zero (0)
    timer.Start();
    checkCudaErrors(hipMemset(d_vectorZero , 0 , sizeNet.sizeVectorNeuron));
    timer.Stop();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,BOOT);

    // copy from device to host
    checkCudaErrors(hipMemcpy(neuralSenses[ SIGHT ].vectorNeuron, d_vectorZero, sizeNet.sizeVectorNeuron, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(neuralSenses[ SIGHT ].vectorFlags , d_vectorZero, sizeNet.sizevectorFlags , hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(neuralSenses[ SIGHT ].binaryCharacteristic,d_vectorZero,sizeNet.sizeBinaryCharacteristic,hipMemcpyDeviceToHost));

    checkCudaErrors(hipMemcpy(neuralSenses[ HEARING ].vectorNeuron, d_vectorZero, sizeNet.sizeVectorNeuron, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(neuralSenses[ HEARING ].vectorFlags , d_vectorZero, sizeNet.sizevectorFlags , hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(neuralSenses[ HEARING ].binaryCharacteristic,d_vectorZero,sizeNet.sizeBinaryCharacteristic,hipMemcpyDeviceToHost));

    // Free memory on device Reserved
    checkCudaErrors(hipFree(d_vectorZero));
}

extern "C"
stateNeuralNetwork recognize(NeuralNetwork * neuralSenses, const SizeNet & sizeNet,
                             unsigned char * h_pattern, Interface * interface, Statistic *& statistic, unsigned char * neuronOrder)
{
    int * d_countHit;
    unsigned char * d_arrayCategory,*d_idsNeuron;
    unsigned char * d_vectorNeuron,* d_vectorFlags,*d_pattern,*d_ptr;
    stateNeuralNetwork state;

    dim3 blockSize (SIZE_CHARACTERISTIC);
    dim3 gridSize  ( (*neuralSenses->ptr) +1 );
    GpuTimer timer;
    Lock lock;

    *(interface->hits) = 0;

    // It allocates memory on the device
    checkCudaErrors(hipMalloc( &d_vectorNeuron, sizeNet.sizeVectorNeuron) );
    checkCudaErrors(hipMalloc( &d_vectorFlags , sizeNet.sizevectorFlags ) );
    checkCudaErrors(hipMalloc( &d_pattern     , sizeof(unsigned char) * SIZE_CHARACTERISTIC));
    checkCudaErrors(hipMalloc( &d_arrayCategory  , sizeof(unsigned char) * (*(neuralSenses->ptr))));
    checkCudaErrors(hipMalloc( &d_idsNeuron   , sizeof(unsigned char) * (*(neuralSenses->ptr))));
    checkCudaErrors(hipMalloc( &d_ptr         , sizeof(unsigned char)));
    checkCudaErrors(hipMalloc( &d_countHit    , sizeof(int)));

    // copy from host to device
    checkCudaErrors( hipMemcpy( d_vectorNeuron, neuralSenses->vectorNeuron ,sizeNet.sizeVectorNeuron, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_vectorFlags , neuralSenses->vectorFlags  ,sizeNet.sizevectorFlags , hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_pattern     , h_pattern                  ,sizeof(unsigned char)*SIZE_CHARACTERISTIC, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_ptr         , neuralSenses->ptr          ,sizeof(unsigned char)   , hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_countHit    , interface->hits            ,sizeof(int)             , hipMemcpyHostToDevice ) );

    //call kernel reconize
    timer.Start();
    recognize<<<gridSize,blockSize>>>(d_vectorNeuron,d_vectorFlags,d_pattern,d_countHit,d_ptr,d_arrayCategory,d_idsNeuron,lock);
    timer.Stop();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,RECOGNIZE);

    // copy from device to host
    checkCudaErrors( hipMemcpy( interface->hits, d_countHit, sizeof(int), hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipMemcpy( neuralSenses->vectorNeuron , d_vectorNeuron, sizeNet.sizeVectorNeuron, hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipMemcpy( neuralSenses->vectorFlags  , d_vectorFlags , sizeNet.sizevectorFlags , hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipMemcpy( neuralSenses->ptr          , d_ptr         , sizeof(unsigned char)   , hipMemcpyDeviceToHost ) );

    interface->freeMem();
    interface->setHit();
    checkCudaErrors(hipMemcpy(interface->arrayCategory,d_arrayCategory ,sizeof(unsigned char)*(* (interface->hits)),hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(interface->id        ,d_idsNeuron  ,sizeof(unsigned char)*(* (interface->hits)),hipMemcpyDeviceToHost));

    if(* (interface->hits) > 1){
        if(* (interface->hits) == 2)
            state = equal(interface->arrayCategory[0],interface->arrayCategory[1])? IS_HIT : DIFF;
        else
            state = compare(interface->arrayCategory,* (interface->hits)) ? IS_HIT : DIFF;
    }
    else if(* (interface->hits) == 1)
        state=IS_HIT;
    else
        state=NO_HIT;

    // Free memory on device Reserved
    checkCudaErrors(hipFree(d_vectorNeuron));
    checkCudaErrors(hipFree(d_vectorFlags));
    checkCudaErrors(hipFree(d_pattern));
    checkCudaErrors(hipFree(d_countHit));
    checkCudaErrors(hipFree(d_ptr));
    checkCudaErrors(hipFree(d_arrayCategory));
    checkCudaErrors(hipFree(d_idsNeuron));
    lock.freeMem();

    return state;
}

extern "C"
void correct(NeuralNetwork * neuralSenses , const SizeNet & sizeNet,
             unsigned char   desiredOutput, int maxThreadsPerBlock, Statistic *&statistic){

    unsigned char * d_desiredOutput;
    unsigned char * d_vectorFlags,* d_ptr;

    dim3 blockSize (maxThreadsPerBlock);
    int numblock= (*(neuralSenses->ptr) % maxThreadsPerBlock == 0) ?
                *(neuralSenses->ptr) / maxThreadsPerBlock:
                *(neuralSenses->ptr) / maxThreadsPerBlock + 1;
    dim3 gridSize(numblock);
    GpuTimer timer;

    // It allocates memory on the device
    checkCudaErrors(hipMalloc(&d_vectorFlags  ,sizeof(unsigned char) * SIZE_FLAGS* (*neuralSenses->ptr)));
    checkCudaErrors(hipMalloc(&d_desiredOutput,sizeof(unsigned char)));
    checkCudaErrors(hipMalloc(&d_ptr,sizeof(unsigned char)));

    // copy from host to device
    checkCudaErrors( hipMemcpy( d_vectorFlags  , neuralSenses->vectorFlags  ,sizeof(unsigned char) * SIZE_FLAGS * (*neuralSenses->ptr), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_ptr          , neuralSenses->ptr          ,sizeof(unsigned char)   , hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_desiredOutput, &desiredOutput             ,sizeof(unsigned char)   , hipMemcpyHostToDevice ) );

    timer.Start();
    //call kernel reconize
    correct<<<gridSize,blockSize>>>(d_vectorFlags,d_ptr,d_desiredOutput);
    timer.Stop();
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,CORRECT);

    // copy from device to host
    checkCudaErrors( hipMemcpy( neuralSenses->vectorFlags , d_vectorFlags , sizeof(unsigned char) * SIZE_FLAGS *(*neuralSenses->ptr), hipMemcpyDeviceToHost ) );

    // Free memory on device Reserved

    checkCudaErrors(hipFree(d_vectorFlags));
    checkCudaErrors(hipFree(d_ptr));
    checkCudaErrors(hipFree(d_desiredOutput));
}

extern "C"
void reset(NeuralNetwork * neuralSenses , const SizeNet & sizeNet, int maxThreadsPerBlock, Statistic *&statistic)
{
    unsigned char * d_vectorFlags,* d_ptr;

    dim3 blockSize (maxThreadsPerBlock);
    int numblock = (*(neuralSenses->ptr) % maxThreadsPerBlock == 0) ?
                *(neuralSenses->ptr) / maxThreadsPerBlock:
                *(neuralSenses->ptr) / maxThreadsPerBlock + 1;
    dim3 gridSize(numblock);
    GpuTimer timer;

    // It allocates memory on the device
    checkCudaErrors(hipMalloc(&d_vectorFlags  ,sizeof(unsigned char) * SIZE_FLAGS * (*neuralSenses->ptr)));
    checkCudaErrors(hipMalloc(&d_ptr,sizeof(unsigned char)));

    // copy from host to device
    checkCudaErrors( hipMemcpy( d_vectorFlags  , neuralSenses->vectorFlags  ,sizeof(unsigned char) * SIZE_FLAGS * (*neuralSenses->ptr), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_ptr          , neuralSenses->ptr          ,sizeof(unsigned char)   , hipMemcpyHostToDevice ) );

    timer.Start();
    //call kernel reconize
    reset<<<gridSize,blockSize>>>(d_vectorFlags,d_ptr);
    timer.Stop();
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,RESET);

    // copy from device to host
    checkCudaErrors( hipMemcpy( neuralSenses->vectorFlags , d_vectorFlags , sizeof(unsigned char) * SIZE_FLAGS * (*neuralSenses->ptr), hipMemcpyDeviceToHost ) );

    // Free memory on device Reserved

    checkCudaErrors(hipFree(d_vectorFlags));
    checkCudaErrors(hipFree(d_ptr));
}

extern "C"
void safeRelation(NeuralNetwork *  neuralSenses, const SizeNet & sizeNet, RelationNetwork relationSenses,
                  Statistic * & statistic, Interface * interface, stateNeuralNetwork *stateSense, unsigned char *whisCategory){
    int * d_countHit;
    stateNeuralNetwork *d_stateSense;
    unsigned char *d_idsNeuron, *d_whisCategory;
    unsigned char * d_vectorFlags;
    unsigned char * d_vectorEar, *d_vectorSigth;


    dim3 blockSize (SIZE_CHARACTERISTIC);
    dim3 gridSize  ( (*neuralSenses->ptr) +1 );
    GpuTimer timer;


    // It allocates memory on the device
    checkCudaErrors(hipMalloc( &d_vectorFlags , sizeNet.sizevectorFlags ) );
    checkCudaErrors(hipMalloc( &d_idsNeuron   , sizeof(unsigned char) * (*(neuralSenses->ptr))));
    checkCudaErrors(hipMalloc( &d_countHit    , sizeof(int)));
    checkCudaErrors(hipMalloc( &d_stateSense    , sizeof(int)));
    checkCudaErrors(hipMalloc( &d_vectorEar, sizeof(unsigned char) * relationSenses.sizeRelationNet) );
    checkCudaErrors(hipMalloc( &d_vectorSigth, sizeof(unsigned char) * relationSenses.sizeRelationNet) );

    // copy from host to device
    checkCudaErrors( hipMemcpy( d_vectorFlags , neuralSenses->vectorFlags  ,sizeNet.sizevectorFlags , hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_whisCategory, whisCategory               ,sizeof(unsigned char)   , hipMemcpyHostToDevice ) );

    checkCudaErrors( hipMemcpy( d_idsNeuron, interface->id,*interface->hits, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_countHit    , interface->hits            , sizeof(int)             , hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_stateSense     , stateSense                    , sizeof (int), hipMemcpyHostToDevice));
    //checkCudaErrors( hipMemcpy( d_sizeRelationNet, relationSenses.sizeRelationNet, sizeof (int), hipMemcpyHostToDevice));

    checkCudaErrors( hipMemcpy( d_vectorEar , relationSenses.vectorEar ,relationSenses.sizeRelationNet, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_vectorSigth , relationSenses.vectorSight ,relationSenses.sizeRelationNet, hipMemcpyHostToDevice ) );

    //call kernel saveRelation
    timer.Start();
    safeRelation<<<gridSize,blockSize>>>(d_vectorFlags,d_countHit,d_idsNeuron, d_vectorEar, d_vectorSigth, d_stateSense, d_whisCategory);
    timer.Stop();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,SAFERELATION);

    // copy from device to host
    checkCudaErrors( hipMemcpy( interface->hits, d_countHit, sizeof(int), hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipMemcpy( neuralSenses->vectorFlags  , d_vectorFlags , sizeNet.sizevectorFlags , hipMemcpyDeviceToHost ) );

    interface->freeMem();

    checkCudaErrors(hipMemcpy(interface->id        ,d_idsNeuron  ,sizeof(unsigned char)*(* (interface->hits)),hipMemcpyDeviceToHost));

    checkCudaErrors( hipMemcpy( relationSenses.vectorEar , d_vectorEar, relationSenses.sizeRelationNet, hipMemcpyDeviceToHost ) );
    checkCudaErrors( hipMemcpy( relationSenses.vectorSight , d_vectorSigth, relationSenses.sizeRelationNet, hipMemcpyDeviceToHost ) );
    //checkCudaErrors( hipMemcpy( relationSenses.sizeRelationNet, d_sizeRelationNet, sizeof(int), hipMemcpyDeviceToHost ) );

    // Free memory on device Reserved
    checkCudaErrors(hipFree(d_vectorFlags));
    checkCudaErrors(hipFree(d_countHit));

    checkCudaErrors(hipFree(d_whisCategory));

    checkCudaErrors(hipFree(d_idsNeuron));

    checkCudaErrors(hipFree(d_vectorEar));
    checkCudaErrors(hipFree(d_vectorSigth));

}

extern "C"
int findOrderNeuron(OrderNetwork * orderNet,const SizeNet & sizeNet,  unsigned char sightID) {

    stateOrderNetwork *d_stateOrder;

    unsigned char * d_relationNeuron;
    unsigned char * d_sightID;
    int *d_minidx;
    int numOrder;

    dim3 blockSize (SIZE_CHARACTERISTIC);
    dim3 gridSize  ( (*orderNet->numRelation) +1 );
    GpuTimer timer;

    //ALLOCATE MEMORY
    checkCudaErrors(hipMalloc( &d_relationNeuron , sizeof(unsigned char) * (*(orderNet->numRelation)) ) );
    checkCudaErrors(hipMalloc( &d_sightID , sizeof(unsigned char) * (*(orderNet->numRelation))  ) );
    checkCudaErrors(hipMalloc( &d_minidx , sizeof(int) ) );

    //HOST TO DEVICE
    checkCudaErrors( hipMemcpy( d_relationNeuron, orderNet->numRelation , sizeof(unsigned char), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_sightID, &sightID , sizeof(unsigned char), hipMemcpyHostToDevice ) );

    timer.Start();
    findOrderNeuron<<<gridSize,blockSize>>>(d_relationNeuron, d_sightID, d_minidx);
    timer.Stop();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    //DEVICE TO HOST
    checkCudaErrors( hipMemcpy( &numOrder , d_minidx, sizeof(int), hipMemcpyDeviceToHost ) );

    checkCudaErrors(hipFree(d_minidx));
    checkCudaErrors(hipFree(d_relationNeuron));

    checkCudaErrors(hipFree(d_sightID));

    return numOrder;
}


/*extern "C"
void newItemCulturalNet(CulturalNet * addNet, int protocol, int LPA, int LPT ){

    unsigned char * d_data, *d_valve, *trueKNW,  *d_LPA, *d_LPT, *d_newData;
    int d_protocol;

    dim3 blockSize (1);
    dim3 gridSize  ( 100);
    GpuTimer timer;


    // It allocates memory on the device
    checkCudaErrors(hipMalloc( &d_data   , sizeof(unsigned char) * (*(neuralSenses->ptr))));
    checkCudaErrors(hipMalloc( &d_data , sizeNet.sizevectorFlags ) );


    // copy from host to device
    checkCudaErrors( hipMemcpy( d_vectorFlags , neuralSenses->vectorFlags  ,sizeNet.sizevectorFlags , hipMemcpyHostToDevice ) );


    //call kernel saveRelation
    timer.Start();
    newItemCulturalNet(<<< * d_data, *d_valve, trueKNW, d_protocol, d_LPA, d_LPT, d_newData>>>);
    timer.Stop();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    calculateStatistic(timer.Elapsed(),statistic,SAFERELATION);

    // copy from device to host
    checkCudaErrors( hipMemcpy( interface->hits, d_countHit, sizeof(int), hipMemcpyDeviceToHost ) );


    // Free memory on device Reserved
    checkCudaErrors(hipFree(d_vectorFlags));

}*/


// methods
template<class T>
bool equal(T a, T b){
    return (a==b)?  true : false;
}

template<class T>
bool compare(T array[], int sizeArray)
{
    T element=array[0];
    for (register int i = 1; i < sizeArray; i++) {

        if(!equal(element,array[i]))
            return false;
    }

    return true;
}

//cuda kernel
__global__ void  safeRelation(unsigned char *d_vectorFlags,
                              int *d_countHit,
                              unsigned char *d_idsNeuron,  unsigned char *d_vectorEar,
                              unsigned char *d_vectorSigth, stateNeuralNetwork *stateSense, unsigned char * d_whisCategory)
{

    int flagIndex   = threadIdx.x + SIZE_FLAGS  * blockIdx.x;
    if(*stateSense == DIFF){
        for(int i = 0; i < *d_countHit; i++){
            if (blockIdx.x == d_idsNeuron[i]){
                d_vectorFlags[flagIndex + CAT] = *d_whisCategory;
            }
        }
        if(threadIdx.x == 0){
            for(int j = 0;j<*d_countHit; j++){
                if ( d_vectorEar[blockIdx.x] == d_idsNeuron[j] ){
                    d_vectorSigth[blockIdx.x] = *d_whisCategory;
                }
            }
        }
    }


}
__global__ void  recognize(unsigned char * d_vectorNeuron, unsigned char *d_vectorFlags,
                           unsigned char *d_pattern, int *d_countHit, unsigned char *d_ptr,
                           unsigned char *d_arrayCategory, unsigned char *d_idsNeuron, Lock lock){

    __shared__ unsigned char sharedVectorNeuron     [SIZE_CHARACTERISTIC];
    __shared__ unsigned char sharedVectorFlags      [SIZE_FLAGS];
    __shared__ unsigned char sharedPattern          [SIZE_CHARACTERISTIC];
    __shared__ int           sharedDistanceManhattan[SIZE_CHARACTERISTIC];

    int vectorIndex = threadIdx.x + SIZE_CHARACTERISTIC * blockIdx.x;
    int flagIndex   = threadIdx.x + SIZE_FLAGS  * blockIdx.x;
    int threadIndex = threadIdx.x;
    unsigned char ptr= *d_ptr;

    sharedVectorNeuron[threadIndex] = d_vectorNeuron [vectorIndex];
    sharedPattern     [threadIndex] = d_pattern      [threadIndex];

    if(threadIndex < SIZE_FLAGS)
        sharedVectorFlags[threadIndex]=d_vectorFlags[flagIndex];

    __syncthreads();            // make sure entire block is loaded!

    if(blockIdx.x == ptr)       //si estoy en la neurona lista para aprender copio el patron
    {
        d_vectorNeuron[vectorIndex]           = sharedPattern[threadIndex];

        if(threadIndex == 0)
            d_vectorFlags [ptr * SIZE_FLAGS + RAT]= RATIO;
    }

    else if(sharedVectorFlags[KNW] == 1 && sharedVectorFlags[DGR]==0)
    {
        sharedDistanceManhattan[threadIndex]= fabsf(sharedPattern[threadIndex]-sharedVectorNeuron[threadIndex]);
        __syncthreads();

        // do reduction in shared mem
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (threadIndex < s)
                sharedDistanceManhattan[threadIndex]+= sharedDistanceManhattan[threadIndex+s];

            __syncthreads();        // make sure all adds at one stage are done!
        }

        // only thread 0 writes result for this block back to global mem
        if (threadIndex == 0)
        {
            if(sharedDistanceManhattan[0] < sharedVectorFlags[RAT])
            {
                d_vectorFlags[SIZE_FLAGS  * blockIdx.x + DIS] = sharedDistanceManhattan[0];
                d_vectorFlags[SIZE_FLAGS  * blockIdx.x + HIT] = 1;

                lock.lock();

                d_arrayCategory [*d_countHit] = sharedVectorFlags[CAT];
                d_idsNeuron  [*d_countHit] = blockIdx.x;
                (*d_countHit)++;

                lock.unlock();

            }
        }
    }
}

__global__ void correct(unsigned char *d_vectorFlags, unsigned char *d_ptr, unsigned char *d_desiredOutput)
{
    int indexGlobal=threadIdx.x + blockDim.x * blockIdx.x;

    unsigned char ratio,hit,dis,category;

    if(indexGlobal < *d_ptr)
    {
        hit   = d_vectorFlags[indexGlobal * SIZE_FLAGS + HIT];
        dis   = d_vectorFlags[indexGlobal * SIZE_FLAGS + DIS];
        category   = d_vectorFlags[indexGlobal * SIZE_FLAGS + CAT];
        ratio = d_vectorFlags[indexGlobal * SIZE_FLAGS + RAT];

        if(hit==1 && category != *d_desiredOutput)
        {
            if(ratio > dis) //NECESARIO?
                d_vectorFlags[ indexGlobal * SIZE_FLAGS + RAT ] = dis;

            if(ratio < MIN_RATIO)
                d_vectorFlags[ indexGlobal * SIZE_FLAGS + DGR ] = 1;
        }
    }
}

__global__ void reset(unsigned char *d_vectorFlags, unsigned char *d_ptr)
{
    int indexGlobal=threadIdx.x + blockDim.x * blockIdx.x;

    if(indexGlobal < *d_ptr)
        d_vectorFlags[ indexGlobal * SIZE_FLAGS + HIT ] = 0;
}

void calculateStatistic(const float &currentTime, Statistic *&statistic, kernels kernel)
{
    statistic[kernel].numExecutions++;
    statistic[kernel].accumulateTime += currentTime;

    if(statistic[kernel].minTime >currentTime)
        statistic[kernel].minTime = currentTime;

    if(statistic [kernel].maxTime < currentTime)
        statistic[kernel].maxTime =currentTime;
}

__global__ void newItemCulturalNet(unsigned char * d_data, bool *d_valve, bool trueKNW, int d_protocol, int d_LPA, int d_LPT, unsigned char d_newData)
{
    int columna = 0;
    int level = 0;
    level = blockIdx.x % 5;
    columna = blockIdx.x/TAMWORD;
    if (d_LPA == columna){
        if(d_LPT == level){
            d_data[blockIdx.x] = d_newData;
            d_valve[blockIdx.x] = true;
        }
    }
}

__global__ void findOrderNeuron(unsigned char *d_relationNeuron, unsigned char * sightID, int *min_idx)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(d_relationNeuron[idx] == sightID[0])
        atomicMin(min_idx, idx);
}

void debugTimer(GpuTimer timer){

    int err = printf("\n%f msecs.\n", timer.Elapsed());

    if (err < 0) {
        //Couldn't print! Probably closed stdout - bad news
        std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
        exit(1);
    }
}
